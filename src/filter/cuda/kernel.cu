#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "stdio.h"
__device__ float4 convolve(int i, int j, const float *kernel, int k, const uint32* img, int imgWidth, int imgHeight) {
	float4 res = make_float4(0.f, 0.f, 0.f, 1.f);

	for (int u = -k; u <= k; u++) {
		for (int v = -k; v <= k; v++) {
			const int ix = clamp(i - u, 0, imgWidth -1);
			const int iy = clamp(j - v, 0, imgHeight - 1);
			const float4 col = toFloat4(img[imgWidth * iy + ix]);
			res +=  col * kernel[(u+k)*(2*k+1) + (v+k)];
		}
	}
	return res;
}

__global__ void convolutionKernel(hipSurfaceObject_t surface, const uint32* srcBuff, int imgWidth, int imgHeight, const float *convKernel, int nbhd) {
	// global thread indices for x and y
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix > imgWidth || iy > imgHeight) {
		return;
	}
	uint32 result = 0;
	if (nbhd > MAX_KERNEL_RADIUS) {
		float4 col = convolve(ix, iy, convKernel, nbhd, srcBuff, imgWidth, imgHeight);
		col = clamp(col, 0.f, 1.f);
		result = toInt(col);
	} else {
		result = convolveShared(srcBuff, imgWidth, imgHeight, convKernel, nbhd);
	}
	if(ix == 1 && iy == 1) {
		uint32 red   = (result >> 24) & 0xFF;
		uint32 green = (result >> 16) & 0xFF;
		uint32 blue  = (result >> 8) & 0xFF;
		uint32 alpha =  result & 0xFF;
		printf("red = %u, green = %u, blue = %u, alpha = %u, \n", red, green, blue, alpha);
	}
	surf2Dwrite(result, surface, ix * sizeof(uint32), iy, hipBoundaryModeClamp);
}

extern "C"
void runCudaKernel(hipSurfaceObject_t glBuffer, void* deviceBuffer, int width, int height, void *convKernel, int nbhd) {
	dim3 threads(TILE_DIM, TILE_DIM);
	dim3 grid(divUp(width, TILE_DIM), divUp(height, TILE_DIM));

	convolutionKernel<<<grid, threads>>>((hipSurfaceObject_t)glBuffer, (uint32*)deviceBuffer, width, height, (float*)convKernel, nbhd);
}

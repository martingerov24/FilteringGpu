#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "stdio.h"

__global__ void convolutionKernel(hipSurfaceObject_t surface, uint32* srcBuff, int imgWidth, int imgHeight, const float *convKernel, int nbhd) {
	// global thread indices for x and y
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix > imgWidth || iy > imgHeight) {
		return;
	}
	uint32 result_pixel = 0;
	float4 result_convolve;
	if (nbhd > MAX_KERNEL_RADIUS) {
		result_convolve = convolve(ix, iy, convKernel, nbhd, srcBuff, imgWidth, imgHeight);
	} else {
		result_convolve = convolveShared(srcBuff, imgWidth, imgHeight, convKernel, nbhd);
	}
	result_convolve = clamp(result_convolve, 0.f, 1.f);
	result_pixel = toInt(result_convolve);
	surf2Dwrite(result_pixel, surface, ix * sizeof(uint32), iy, hipBoundaryModeZero);
}

extern "C"
void runCudaKernel(hipSurfaceObject_t glBuffer, void* deviceBuffer, int width, int height, void *convKernel, int nbhd) {
	dim3 threads(TILE_DIM, TILE_DIM);
	dim3 grid(divUp(width, TILE_DIM), divUp(height, TILE_DIM));

	convolutionKernel<<<grid, threads>>>((hipSurfaceObject_t)glBuffer, (uint32*)deviceBuffer, width, height, (float*)convKernel, nbhd);
}
